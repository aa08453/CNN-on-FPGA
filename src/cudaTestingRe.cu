#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main()
{
    int N = 1 << 20; // 1 million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize host input vectors
    for (int i = 0; i < N; ++i)
    {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }

    // Allocate device memory
    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    bool success = true;
    for (int i = 0; i < N; ++i)
    {
        if (h_C[i] != h_A[i] + h_B[i])
        {
            std::cerr << "Mismatch at index " << i << ": "
                << h_C[i] << " != " << h_A[i] + h_B[i] << std::endl;
            success = false;
            break;
        }
    }

    std::cout << (success ? "Success! All values match." : "Failure.") << std::endl;

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
