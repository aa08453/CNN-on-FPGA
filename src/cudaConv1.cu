#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "cnpy.h"
#include <vector>
#include <cmath>

#include <chrono>

using namespace std;

__global__ void conv2dKernel(
    const float* input,
    const float* weights,
    float* output,
    int N, int H, int W_in, int C_in,
    int K, int C_out, int H_out, int W_out)
{
    int n = blockIdx.z / ((C_out + 3) / 4);
    int co = (blockIdx.z % ((C_out + 3) / 4)) * blockDim.z + threadIdx.z;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int w = blockIdx.x * blockDim.x + threadIdx.x;

    if (n < N && h < H_out && w < W_out && co < C_out) {
        float sum = 0.0f;
        for (int kh = 0; kh < K; ++kh)
            for (int kw = 0; kw < K; ++kw)
                for (int ci = 0; ci < C_in; ++ci) {
                    int in_h = h + kh;
                    int in_w = w + kw;
                    int inputIdx = ((n * H + in_h) * W_in + in_w) * C_in + ci;
                    int weightIdx = ((kh * K + kw) * C_in + ci) * C_out + co;
                    sum += input[inputIdx] * weights[weightIdx];
                }
        int outIdx = ((n * H_out + h) * W_out + w) * C_out + co;
        output[outIdx] = sum;
    }
}

bool compare2(const float* output, const float* expected, size_t outSize, float eps = 1e-3f) {
    for (size_t n = 0; n < outSize; ++n) {
        float diff = fabs(output[n] - expected[n]);
        if (diff > eps)
        {
            cout << "Mismatch" << endl;
            return false;
        }
    }
    return true;
}

int main() {
    // Load data
    auto Z_array = cnpy::npy_load("input_image.npy");
    auto W_array = cnpy::npy_load("weights.npy");
    auto expected_array = cnpy::npy_load("expected_output.npy");

    float* Z2 = Z_array.data<float>();
    float* weight2 = W_array.data<float>();
    float* expected = expected_array.data<float>();

    vector<size_t> inpShape = Z_array.shape;
    vector<size_t> weightShape = W_array.shape;

    int N = inpShape[0], H = inpShape[1], W_in = inpShape[2], C_in = inpShape[3];
    int K = weightShape[0], C_out = weightShape[3];
    int H_out = H - K + 1;
    int W_out = W_in - K + 1;

    size_t Z_size = N * H * W_in * C_in;
    size_t W_size = K * K * C_in * C_out;
    size_t outSize = N * H_out * W_out * C_out;


    cout << "starting cuda testing" << endl;
    float* d_input, * d_weight, * d_output;
    size_t inputSize = N * H * W_in * C_in * sizeof(float);
    size_t weightSize = K * K * C_in * C_out * sizeof(float);
    size_t outputSize = N * H_out * W_out * C_out * sizeof(float);

    // Allocate GPU memory
    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_weight, weightSize);
    hipMalloc(&d_output, outputSize);

    // Copy data to GPU
    hipMemcpy(d_input, Z2, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight2, weightSize, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((W_out + 15) / 16, (H_out + 15) / 16, N);

    auto start_cuda = std::chrono::high_resolution_clock::now();
    conv2dKernel << <gridDim, blockDim >> > (
        d_input, d_weight, d_output,
        N, H, W_in, C_in,
        K, C_out,
        H_out, W_out
        );
    hipDeviceSynchronize();
    auto end_cuda = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_cuda = end_cuda - start_cuda;
    std::cout << "CUDA convolution took " << elapsed_cuda.count() << " seconds.\n";

    // Copy back result
    std::vector<float> cuda_output(N * H_out * W_out * C_out);
    hipMemcpy(cuda_output.data(), d_output, outputSize, hipMemcpyDeviceToHost);

    // Compare
    if (compare2(cuda_output.data(), expected_array.data<float>(), N * H_out * W_out * C_out))
        cout << "CUDA output matches expected NumPy result!\n";
    else
        cout << "CUDA output does not match.\n";

    // Cleanup
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);

}